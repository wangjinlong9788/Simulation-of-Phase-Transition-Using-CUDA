#include "hip/hip_runtime.h"
/*

 
Use "REAL" as floating point type instead of double or float
 
Compile with optional flag:
    -DDOUBLE to use double instead of  float
     requires also -arch sm_21
 
 
single precision code:
 
> nvcc template.cu fileutils.cpp stringutils.cpp graphicstools.cpp -lcufft -o demo_single
 
double precision code:
 
> nvcc template.cu fileutils.cpp stringutils.cpp graphicstools.cpp -lcufft -DDOUBLE -arch sm_21 -o demo_single

 
*/

#include <stdio.h>
#include <stdlib.h> /* for rand() */
#include <unistd.h> /* for getpid() */
#include <time.h> /* for time() */
#include <math.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "fileutils.h"
#include "stringutils.h"
#include "graphicstools.h"

// ******************************************************

#define PI	3.1415926535897932384626433832795
#define TWOPI 6.28318530717958647692528676655901

// construct REAL "type," depending on desired precision
// set the maximum number of threads

#ifdef DOUBLE
 #define REAL double
 #define MAXT 256
#else
 #define REAL float
 #define MAXT 512
#endif

typedef struct {
	REAL re;
	REAL im;
} COMPLEX;

// ******************************************************
//calculate the k-index in order to determine the correct k-vector for a given x,y,z-index

#define k_INDEX(i,L) ((i)<=((L)/2)?(i):((i)-(L)))

// ******************************************************

//initialize a real GPU array with a constant
__global__ void G_setrealconst(int N,REAL *a,REAL val) {
    int idx=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(idx<N) a[idx]=val;
};


//multiply two complex GPU arrays (A,B) and store result in A
__global__ void G_mulcarray(int N,COMPLEX *A,COMPLEX *B)
{
    int i=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    REAL re,im,re2,im2;
    if(i<N)
    {
        re=A[i].re;im=A[i].im;
        re2=B[i].re;im2=B[i].im;
        A[i].re=re*re2-im*im2;
        A[i].im=im*re2+re*im2;
    }
};


// ******************************************************


//execute the FFT on the GPU, zin and zout can be the same array for "in-place" FFT (a little slower)
//set "fwd" to false for inverse FFT
void G_FFT(COMPLEX *zin,COMPLEX *zout,hipfftHandle &fftPlan,bool fwd=true)
{
#ifdef DOUBLE
    if(fwd) hipfftExecZ2Z(fftPlan,(hipfftDoubleComplex*) zin,(hipfftDoubleComplex*) zout,HIPFFT_FORWARD);
    else    hipfftExecZ2Z(fftPlan,(hipfftDoubleComplex*) zin,(hipfftDoubleComplex*) zout,HIPFFT_BACKWARD);
#else
    if(fwd) hipfftExecC2C(fftPlan,(hipfftComplex*) zin,(hipfftComplex*) zout,HIPFFT_FORWARD);
    else    hipfftExecC2C(fftPlan,(hipfftComplex*) zin,(hipfftComplex*) zout,HIPFFT_BACKWARD);
#endif
};

 
// ******************************************************

//split a complex array in two real arrays containing amplitude^2 and phase
__global__ void G_ampphase(int N,COMPLEX *A,REAL* amp2,REAL* phase)
{
    int idx=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    REAL re,im;
    if(idx<N)
    {
        re=A[idx].re;im=A[idx].im;
        amp2[idx]=re*re+im*im;
        phase[idx]=atan2(im,re);
    }
};

//split a complex array in two real arrays containing real and imaginary parts
__global__ void G_splitreim(int N,COMPLEX *A,REAL* re,REAL* im)
{
    int idx=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    if(idx<N)
    {
        im[idx]=A[idx].im;
        re[idx]=A[idx].re;
    }
};

// ******************************************************


//check for a CUDA error, use argument for identification
bool cerr(const char *s="n/a")
{
    hipError_t err=hipGetLastError();
    if(err==hipSuccess)
        return false;
    printf("CUDA error [%s]: %s\n",s,hipGetErrorString(err));
    return true;
};


//some function initializing a 2D complex array
__global__ void G_function(int Nx,int Ny, COMPLEX *f,REAL t) {
    int idx=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    int i,j;
    if(idx<Nx*Ny) {
        i=idx%Nx;
        j=idx/Nx;
        
        f[idx].re=sin(0.1*i+t)*cos(0.1*t*j);
        f[idx].im=-sin(0.1*j+t)*cos(0.1*t*i);
    }
};


//function to calculate the x-derivate in Fourier space
__global__ void G_kernel(int Nx, COMPLEX *f,REAL dkx) {
    int idx=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
    int i,j,ki;
    REAL x,y,k;
    if(idx<Nx*Nx) {
        i=idx%Nx;
        j=idx/Nx;
        
        //calculate the x-derivative in Fourier space
        ki=k_INDEX(i,Nx); //the Fourier component
        //kj=k_INDEX(j,Ny);
        
        //multipy i*k_x to f(k_x,k_y)
        k=dkx*ki;
        y=k*f[idx].re;
        x=-k*f[idx].im;
        
        f[idx].re=x;
        f[idx].im=y;
    }
}

// ******************************************************

//outputs an NetPBM image based on a real array a, m is the minimum value of a and M the maximum, nx&ny the dimension of a and cgrad a color gradient
/* cgrad values
 0: rainbow
 1: rainbow 2
 2: rainbow 3
 3: dark rainbow
 4: temperature
 5: temperature 2
 6: thermo
 7: solar
 8: sunset
 9: neon
*/

void writeBM_real(string fn,REAL *a,REAL m,REAL M,int nx,int ny,int cgrad)
{
    int i,n;
    dcolor dcol;
    unsigned int col;
    REAL val,dci=1.0/(M-m);
    unsigned int *rgb;
    unsigned char *gray;
    PXMfile *Ifile;
    colorfunction *colors;
    
    n=nx*ny;
    
    colors= new colorfunction();
    colors->selectgradient(cgrad);
    
    rgb=new unsigned int[n];
    
    gray=(unsigned char *) rgb;
    for(i=0;i<n;i++)
    {
        val=(a[i]-m)*dci;
        if(cgrad<1) {
            col=(unsigned int) (256*val);if(col>255) col=255;
            gray[i]=col;}
        else {
            dcol=colors->getgradientcolor(val);col=colors->get32bitcolor(dcol);
            rgb[i]=col;}
    }
    
    Ifile=new PXMfile(fn,(cgrad<1?PXM_P5:PXM_P6));
    Ifile->setsize(nx,ny);
    if(cgrad<1) Ifile->writefile(gray,nx*ny);
    else Ifile->writefile(rgb,nx*ny);
    delete Ifile;
    
    delete[] rgb;
    delete colors;
};

// ******************************************************

int main(int argc,char *argv[])
{
    int N,i,n,dim;
    int threads,blocks;
    REAL t,dt,dkx,L,mval,Mval,x;
    size_t fmem,tmem;
    COMPLEX *GF,*f,*Gtmp;
    REAL *amp2,*phase,*Gamp2,*Gphase;
    hipfftHandle fftPlan;
    
    //welcome info
    
    printf("template program using ");
#ifdef DOUBLE
    printf("double");
#else
    printf("single");
#endif
    printf(" precision arithmetics.\n");
    
    //default parameters
    //assume square
    dim=2;
    N=256;
    L=256.0;

    // check if arguments are present and read them
    
    if(argc > 1 ) N = atoi(argv[1]);
    
    //excute
    

    hipSetDevice(0);
    
    hipMemGetInfo(&fmem,&tmem);
    printf("GPU memory before allocation free: %lu, total: %lu\n",fmem,tmem);

    threads=MAXT;
    blocks=N*N/threads+(N*N%threads==0?0:1);
    
    
    hipMalloc(&GF,N*N*sizeof(COMPLEX));
    hipMalloc(&Gtmp,N*N*sizeof(COMPLEX));
    hipMalloc(&Gamp2,N*N*sizeof(REAL));
    hipMalloc(&Gphase,N*N*sizeof(REAL));
    f=new COMPLEX[N*N];
    amp2=new REAL[N*N];
    phase=new REAL[N*N];
    
    
    //for FFT
    dkx=TWOPI/L;
    
    //include normalization in dkx:
    dkx=dkx/(1.0*N*N);
    //we need a "plan"
#ifdef DOUBLE
         if(dim==1) hipfftPlan1d(&fftPlan, N, HIPFFT_Z2Z,1);
    else if(dim==2) hipfftPlan2d(&fftPlan, N, N, HIPFFT_Z2Z) ;
    else if(dim==3) hipfftPlan3d(&fftPlan, N, N, N, HIPFFT_Z2Z);
#else
         if(dim==1) hipfftPlan1d(&fftPlan, N, HIPFFT_C2C,1);
    else if(dim==2) hipfftPlan2d(&fftPlan, N, N, HIPFFT_C2C) ;
    else if(dim==3) hipfftPlan3d(&fftPlan, N, N, N, HIPFFT_C2C);
#endif
    cerr("FFT plan"); //check for error
    
    
    t=0.0;dt=0.1;
    for(n=0;n<100;n++) {
        G_function<<<blocks,threads>>>(N,N,GF,t);
        
        
        //output
        G_ampphase<<<blocks,threads>>>(N*N,GF,Gamp2,Gphase);
        hipMemcpy(amp2,Gamp2,N*N*sizeof(REAL),hipMemcpyDeviceToHost);
        hipMemcpy(phase,Gphase,N*N*sizeof(REAL),hipMemcpyDeviceToHost);
        writeBM_real("test_amp2_"+IntToStrF(n,4),amp2,0,2,N,N,5);
        writeBM_real("test_phase_"+IntToStrF(n,4),phase,-PI,PI,N,N,6);
        
        //FFT
        G_FFT(GF,Gtmp,fftPlan); //forward
        G_kernel<<<blocks,threads>>>(N,Gtmp,dkx);
        G_FFT(Gtmp,GF,fftPlan,false); //inverse
        
        //output
        G_splitreim<<<blocks,threads>>>(N*N,GF,Gamp2,Gphase);
        hipMemcpy(amp2,Gamp2,N*N*sizeof(REAL),hipMemcpyDeviceToHost);
        hipMemcpy(phase,Gphase,N*N*sizeof(REAL),hipMemcpyDeviceToHost);
        mval=Mval=amp2[0];
        for(i=1;i<N*N;i++) {x=amp2[i];if(x>Mval) Mval=x;else if(x<mval) mval=x;}
        printf("%f %f; ",mval,Mval);
        writeBM_real("test_dx_re_"+IntToStrF(n,4),amp2,mval,Mval,N,N,5);
        mval=Mval=phase[0];
        for(i=1;i<N*N;i++) {x=phase[i];if(x>Mval) Mval=x;else if(x<mval) mval=x;}
        printf("%f %f\n",mval,Mval);
        writeBM_real("test_dx_im_"+IntToStrF(n,4),phase,mval,Mval,N,N,5);
        
        
        t+=dt;
    }
    
    delete[] f;
    delete[] amp2;
    delete[] phase;
    hipFree(GF);
    hipFree(Gtmp);
    hipFree(Gamp2);
    hipFree(Gphase);

    
    return 0;
    }

// ******************************************************
